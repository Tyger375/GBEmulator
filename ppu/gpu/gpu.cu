#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <SDL3/SDL_stdinc.h>
#include "../../defs.hpp"
#include "gpu.cuh"

__device__ unsigned int mapRGB(unsigned char r, unsigned char g, unsigned char b) {
    // Assume RGB888 format (8 bits per channel)
    return (r << 16) | (g << 8) | b;
}

__device__ Uint32 color_to_rgb(Byte color) {
    switch (color) {
        case 0: {
            return mapRGB(255, 255, 255);
        }
        case 1: {
            return mapRGB(169, 169, 169);
        }
        case 2: {
            return mapRGB(84, 84, 84);
        }
        case 3: {
            return mapRGB(0, 0, 0);
        }
        default: return mapRGB(255, 255, 255);
    }
}

__global__ void cuda_build_video(
        Byte* video,
        Uint32* scaled_video,
        int scale
        ) {
    u32 pos = blockIdx.y * DISPLAY_WIDTH + blockIdx.x;
    u32 scaled_pos = (blockIdx.y * scale + threadIdx.y) * (DISPLAY_WIDTH * scale) + (blockIdx.x * scale + threadIdx.x);
    scaled_video[scaled_pos] = color_to_rgb(video[pos]);
}

void build_video(Byte* video, u32 width, u32 height, Uint32* scaled_video, int scale) {
    u32 new_width = width * scale;
    u32 new_height = height * scale;

    Byte* cuda_video;
    size_t pitch = 0;
    hipMallocPitch((void**)&cuda_video, &pitch, width * sizeof(Byte), height);
    hipMemcpy(
            cuda_video,
            video,
            (width * height) * sizeof(Byte),
            hipMemcpyHostToDevice
            );

    Uint32* cuda_scaled_video;
    size_t scaled_pitch = 0;
    hipMallocPitch((void**)&cuda_scaled_video, &scaled_pitch, new_width * sizeof(Uint32), new_height);

    dim3 video_dim(width, height);
    dim3 scale_dim(scale, scale);
    cuda_build_video<<<video_dim, scale_dim>>>(cuda_video, cuda_scaled_video, scale);

    hipMemcpy(
            scaled_video,
            cuda_scaled_video,
            (new_width * new_height) * sizeof(Uint32),
            hipMemcpyDeviceToHost
    );
}